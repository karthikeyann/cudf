/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <io/fst/type_inference.cuh>
#include <io/utilities/hostdevice_vector.hpp>
#include <io/utilities/trie.cuh>

#include <cudf_test/base_fixture.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>

#include <cstddef>
#include <string>
#include <vector>

using cudf::io::fst::detail::detect_data_type;
using cudf::io::fst::detail::inference_options;

// Base test fixture for tests
struct TypeInference : public cudf::test::BaseFixture {
};

TEST_F(TypeInference, Basic)
{
  auto stream  = rmm::cuda_stream_default;
  auto options = inference_options{};

  options.trie_true  = cudf::detail::create_serialized_trie({"true"}, stream);
  options.trie_false = cudf::detail::create_serialized_trie({"false"}, stream);
  options.trie_na    = cudf::detail::create_serialized_trie({"", "null"}, stream);

  std::string data = "[42,52,5]";
  rmm::device_uvector<char> d_data{data.size(), stream};
  hipMemcpyAsync(
    d_data.data(), data.data(), data.size() * sizeof(char), hipMemcpyHostToDevice, stream.value());

  std::size_t constexpr size = 3;
  rmm::device_uvector<thrust::pair<int32_t, std::size_t>> d_col_strings{size, stream};
  d_col_strings.set_element(0, {1, 2}, stream);
  d_col_strings.set_element(1, {4, 2}, stream);
  d_col_strings.set_element(2, {7, 1}, stream);

  auto res_type = detect_data_type(options.view(), d_data, d_col_strings.begin(), size, stream);

  EXPECT_EQ(res_type, cudf::data_type{cudf::type_id::INT64});
}
