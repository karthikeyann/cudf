#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <benchmarks/fixture/benchmark_fixture.hpp>
#include <benchmarks/synchronization/synchronization.hpp>

#include <cudf_test/column_wrapper.hpp>

#include <cudf/column/column.hpp>
#include <cudf/copying.hpp>

#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/detail/tagged_iterator.h>
#include <thrust/random.h>

// to enable, run cmake with -DBUILD_BENCHMARKS=ON

template <typename Tag, typename Iterator>
inline auto make_tagged_iterator(Iterator iter)
{
  return thrust::detail::tagged_iterator<Iterator, Tag>(iter);
}

template <typename T>
void BM_contiguous_split_common(benchmark::State& state,
                                std::vector<T>& columns,
                                int64_t num_rows,
                                int64_t num_splits,
                                int64_t bytes_total)
{
  // generate splits
  cudf::size_type split_stride = num_rows / num_splits;
  std::vector<cudf::size_type> splits;
  for (int idx = 0; idx < num_rows; idx += split_stride) {
    splits.push_back(std::min(idx + split_stride, static_cast<cudf::size_type>(num_rows)));
  }

  for (auto&& c : columns) {
    // computing the null count is not a part of the benchmark's target code path, and we want the
    // property to be pre-computed so that we measure the performance of only the intended code path
    [[maybe_unused]] auto const nulls = c->null_count();
  }

  cudf::table src_table(std::move(columns));

  for (auto _ : state) {
    cuda_event_timer raii(state, true);  // flush_l2_cache = true, stream = 0
    auto result = cudf::contiguous_split(src_table, splits);
  }

  state.SetBytesProcessed(static_cast<int64_t>(state.iterations()) * bytes_total);
}

class ContiguousSplit : public cudf::benchmark {
};

void BM_contiguous_split(benchmark::State& state)
{
  int64_t total_desired_bytes = state.range(0);
  cudf::size_type num_cols    = state.range(1);
  cudf::size_type num_splits  = state.range(2);
  bool include_validity       = state.range(3) == 0 ? false : true;

  cudf::size_type el_size = 4;  // ints and floats
  int64_t num_rows        = total_desired_bytes / (num_cols * el_size);

  // generate input table
  auto valids = thrust::constant_iterator<bool>(true);
  std::vector<std::unique_ptr<cudf::column>> src_cols(num_cols);
  for (int idx = 0; idx < num_cols; idx++) {
    auto rand_elements = make_tagged_iterator<thrust::device_system_tag>(
      cudf::detail::make_counting_transform_iterator(0u, [idx] __device__(uint32_t i) {
        thrust::default_random_engine rng(31337 + idx);
        thrust::uniform_int_distribution<uint32_t> dist;
        rng.discard(i);
        return dist(rng);
      }));
    if (include_validity) {
      src_cols[idx] =
        cudf::test::fixed_width_column_wrapper<int>(rand_elements, rand_elements + num_rows, valids)
          .release();
    } else {
      src_cols[idx] =
        cudf::test::fixed_width_column_wrapper<int>(rand_elements, rand_elements + num_rows)
          .release();
    }
  }

  size_t total_bytes = total_desired_bytes;
  if (include_validity) { total_bytes += num_rows / (sizeof(cudf::bitmask_type) * 8); }

  BM_contiguous_split_common(state, src_cols, num_rows, num_splits, total_bytes);
}

class ContiguousSplitStrings : public cudf::benchmark {
};

int rand_range(int r)
{
  return static_cast<int>((static_cast<float>(rand()) / static_cast<float>(RAND_MAX)) *
                          (float)(r - 1));
}

void BM_contiguous_split_strings(benchmark::State& state)
{
  int64_t total_desired_bytes = state.range(0);
  cudf::size_type num_cols    = state.range(1);
  cudf::size_type num_splits  = state.range(2);
  bool include_validity       = state.range(3) == 0 ? false : true;
  using string_pair           = thrust::pair<const char*, cudf::size_type>;

  const int64_t string_len = 8;
  cudf::test::strings_column_wrapper w_strings(
    {"aaaaaaaa", "bbbbbbb", "cccccc", "ddddd", "eeee", "fff", "gg", "h", ""},
    {1, 1, 1, 1, 1, 1, 1, 1, 0});
  cudf::column_view d_strings = w_strings;
  int64_t col_len_bytes       = total_desired_bytes / num_cols;
  int64_t num_rows            = col_len_bytes / string_len;

  // generate input table
  std::vector<std::unique_ptr<cudf::column>> src_cols(num_cols);
  for (int64_t idx = 0; idx < num_cols; idx++) {
    // fill in a random set of strings
    auto rand_elements = make_tagged_iterator<thrust::device_system_tag>(
      cudf::detail::make_counting_transform_iterator(
        0u, [idx, sz = d_strings.size() - !include_validity] __device__(uint32_t i) {
          thrust::default_random_engine rng(31337 + idx);
          thrust::uniform_int_distribution<uint32_t> dist{0, sz - 1u};
          rng.discard(i);
          return dist(rng);
        }));
    auto d_elements =
      cudf::test::fixed_width_column_wrapper<int>(rand_elements, rand_elements + num_rows);
    auto d_table = cudf::gather(
      cudf::table_view({d_strings}), d_elements, cudf::out_of_bounds_policy::DONT_CHECK);
    if (!include_validity) d_table->get_column(0).set_null_mask(rmm::device_buffer{}, 0);
    src_cols[idx] = std::move(d_table->release()[0]);
  }

  size_t total_bytes = total_desired_bytes + (num_rows * sizeof(cudf::size_type));
  if (include_validity) { total_bytes += num_rows / (sizeof(cudf::bitmask_type) * 8); }

  BM_contiguous_split_common(state, src_cols, num_rows, num_splits, total_bytes);
}

#define CSBM_BENCHMARK_DEFINE(name, size, num_columns, num_splits, validity) \
  BENCHMARK_DEFINE_F(ContiguousSplit, name)(::benchmark::State & state)      \
  {                                                                          \
    BM_contiguous_split(state);                                              \
  }                                                                          \
  BENCHMARK_REGISTER_F(ContiguousSplit, name)                                \
    ->Args({size, num_columns, num_splits, validity})                        \
    ->Unit(benchmark::kMillisecond)                                          \
    ->UseManualTime()                                                        \
    ->Iterations(8)
CSBM_BENCHMARK_DEFINE(6Gb512ColsNoValidity, (int64_t)6 * 1024 * 1024 * 1024, 512, 256, 0);
CSBM_BENCHMARK_DEFINE(6Gb512ColsValidity, (int64_t)6 * 1024 * 1024 * 1024, 512, 256, 1);
CSBM_BENCHMARK_DEFINE(6Gb10ColsNoValidity, (int64_t)6 * 1024 * 1024 * 1024, 10, 256, 0);
CSBM_BENCHMARK_DEFINE(6Gb10ColsValidity, (int64_t)6 * 1024 * 1024 * 1024, 10, 256, 1);

CSBM_BENCHMARK_DEFINE(4Gb512ColsNoValidity, (int64_t)4 * 1024 * 1024 * 1024, 512, 256, 0);
CSBM_BENCHMARK_DEFINE(4Gb512ColsValidity, (int64_t)4 * 1024 * 1024 * 1024, 512, 256, 1);
CSBM_BENCHMARK_DEFINE(4Gb10ColsNoValidity, (int64_t)4 * 1024 * 1024 * 1024, 10, 256, 0);
CSBM_BENCHMARK_DEFINE(46b10ColsValidity, (int64_t)4 * 1024 * 1024 * 1024, 10, 256, 1);

CSBM_BENCHMARK_DEFINE(1Gb512ColsNoValidity, (int64_t)1 * 1024 * 1024 * 1024, 512, 256, 0);
CSBM_BENCHMARK_DEFINE(1Gb512ColsValidity, (int64_t)1 * 1024 * 1024 * 1024, 512, 256, 1);
CSBM_BENCHMARK_DEFINE(1Gb10ColsNoValidity, (int64_t)1 * 1024 * 1024 * 1024, 10, 256, 0);
CSBM_BENCHMARK_DEFINE(1Gb10ColsValidity, (int64_t)1 * 1024 * 1024 * 1024, 10, 256, 1);

#define CSBM_STRINGS_BENCHMARK_DEFINE(name, size, num_columns, num_splits, validity) \
  BENCHMARK_DEFINE_F(ContiguousSplitStrings, name)(::benchmark::State & state)       \
  {                                                                                  \
    BM_contiguous_split_strings(state);                                              \
  }                                                                                  \
  BENCHMARK_REGISTER_F(ContiguousSplitStrings, name)                                 \
    ->Args({size, num_columns, num_splits, validity})                                \
    ->Unit(benchmark::kMillisecond)                                                  \
    ->UseManualTime()                                                                \
    ->Iterations(8)

CSBM_STRINGS_BENCHMARK_DEFINE(4Gb512ColsNoValidity, (int64_t)4 * 1024 * 1024 * 1024, 512, 256, 0);
CSBM_STRINGS_BENCHMARK_DEFINE(4Gb512ColsValidity, (int64_t)4 * 1024 * 1024 * 1024, 512, 256, 1);
CSBM_STRINGS_BENCHMARK_DEFINE(4Gb10ColsNoValidity, (int64_t)4 * 1024 * 1024 * 1024, 10, 256, 0);
CSBM_STRINGS_BENCHMARK_DEFINE(4Gb10ColsValidity, (int64_t)4 * 1024 * 1024 * 1024, 10, 256, 1);

CSBM_STRINGS_BENCHMARK_DEFINE(1Gb512ColsNoValidity, (int64_t)1 * 1024 * 1024 * 1024, 512, 256, 0);
CSBM_STRINGS_BENCHMARK_DEFINE(1Gb512ColsValidity, (int64_t)1 * 1024 * 1024 * 1024, 512, 256, 1);
CSBM_STRINGS_BENCHMARK_DEFINE(1Gb10ColsNoValidity, (int64_t)1 * 1024 * 1024 * 1024, 10, 256, 0);
CSBM_STRINGS_BENCHMARK_DEFINE(1Gb10ColsValidity, (int64_t)1 * 1024 * 1024 * 1024, 10, 256, 1);
