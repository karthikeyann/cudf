#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <fixture/benchmark_fixture.hpp>
#include <synchronization/synchronization.hpp>
#include <tests/utilities/column_wrapper.hpp>
#include <cudf/detail/aggregation/aggregation.hpp>
#include <cudf/sorting.hpp>
#include <cudf/copying.hpp>
#include <cudf/table/table.hpp>
#include <cudf/compiled_binaryop.hpp>

#include <random>
#include <memory>

class BinOp : public cudf::benchmark {};

// TODO: put it in a struct so `uniform` can be remade with different min, max
template <typename T>
T random_int(T min, T max)
{
  static unsigned seed = 13377331;
  static std::mt19937 engine{seed};
  static std::uniform_int_distribution<T> uniform{min, max};

  return uniform(engine);
}

void BM_basic_sum1(benchmark::State& state){
  using wrapper = cudf::test::fixed_width_column_wrapper<int64_t>;
  //cudf::data_type(cudf::experimental::type_to_id<int64_t>()));
  const cudf::size_type column_size{(cudf::size_type)state.range(0)};
  auto data_it = cudf::test::make_counting_transform_iterator(0,
    [=](cudf::size_type row) { return random_int(0, 100); });
  wrapper keys(data_it, data_it + column_size);
  wrapper vals(data_it, data_it + column_size);
  cudf::column_view lhs = keys;
  cudf::column_view rhs = vals;

  for(auto _ : state) {
    cuda_event_timer timer(state, true);
    auto out1 = cudf::experimental::experimental_binary_operation1(lhs, rhs, lhs.type());
  }
  state.SetComplexityN(state.range(0));
}

void BM_basic_sum2(benchmark::State& state){
  using wrapper = cudf::test::fixed_width_column_wrapper<int64_t>;
  //cudf::data_type(cudf::experimental::type_to_id<int64_t>()));
  const cudf::size_type column_size{(cudf::size_type)state.range(0)};
  auto data_it = cudf::test::make_counting_transform_iterator(0,
    [=](cudf::size_type row) { return random_int(0, 100); });
  wrapper keys(data_it, data_it + column_size);
  wrapper vals(data_it, data_it + column_size);
  cudf::column_view lhs = keys;
  cudf::column_view rhs = vals;

  for(auto _ : state) {
    cuda_event_timer timer(state, true);
    auto out1 = cudf::experimental::experimental_binary_operation2(lhs, rhs, lhs.type());
  }
  state.SetComplexityN(state.range(0));
}


void BM_basic_sum3(benchmark::State& state){
  using wrapper = cudf::test::fixed_width_column_wrapper<int64_t>;
  //cudf::data_type(cudf::experimental::type_to_id<int64_t>()));
  const cudf::size_type column_size{(cudf::size_type)state.range(0)};
  auto data_it = thrust::make_counting_iterator<int64_t>(0);
  //auto data_it = cudf::test::make_counting_transform_iterator(0,
  //  [=](cudf::size_type row) { return random_int(0, 100); });
  wrapper keys(data_it, data_it + column_size);
  wrapper vals(data_it, data_it + column_size);
  cudf::column_view lhs = keys;
  cudf::column_view rhs = vals;

  for(auto _ : state) {
    cuda_event_timer timer(state, true);
    auto out1 = cudf::experimental::experimental_binary_operation3(lhs, rhs, lhs.type());
  }
  state.SetComplexityN(state.range(0));
}

#define BBM_BENCHMARK_DEFINE(BNAME, name)                                      \
  BENCHMARK_DEFINE_F(BinOp, name)(::benchmark::State & state) {                \
    BNAME(state);                                                              \
  }                                                                            \
  BENCHMARK_REGISTER_F(BinOp, name)                                           \
      ->UseManualTime()                                                        \
      ->Unit(benchmark::kMillisecond)                                          \
      ->Arg(1000)       /* 1k  */                                              \
      ->Arg(10000)      /* 10k */                                              \
      ->Arg(100000)     /* 100k*/                                              \
      ->Arg(1000000)    /* 1M  */                                              \
      ->Arg(10000000)   /* 10M */                                              \
      ->Arg(100000000)  /* 100M*/                                              \
      ->Arg(1000000000);/* 1G  */

BBM_BENCHMARK_DEFINE(BM_basic_sum1, exp1);
BBM_BENCHMARK_DEFINE(BM_basic_sum2, exp2);
BBM_BENCHMARK_DEFINE(BM_basic_sum3, exp3);
