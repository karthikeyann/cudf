#include "hip/hip_runtime.h"
//#include <cudf/cudf.h>

#include <thrust/transform_reduce.h>
#include <cudf/utilities/legacy/type_dispatcher.hpp>
#include "reduction_functions.cuh"
#include <bitmask/legacy/legacy_bitmask.hpp>
//#include <rmm/thrust_rmm_allocator.h>

// int any types  -> t=float
// float -> t=float
// double -> t=double
// TODO: works with nan/null? (null ignored)

// TODO need to replace unsigned int with to hold 2^64
typedef thrust::pair<double, unsigned int> Mypair;

template <typename ElementType, typename ResultType>
struct abs_diff : public thrust::unary_function<ElementType, ResultType> {
  const double y;
  ElementType const* elements{};    ///< pointer of cudf data array
  gdf_valid_type const* bitmask{};  ///< pointer of cudf bitmask (null) array
  ResultType const
      identity{};  ///< identity value used when the validity is false

  abs_diff(const gdf_column column, const ResultType _identity, double _y)
      : y(_y),
        elements(static_cast<const ElementType*>(column.data)),
        bitmask(reinterpret_cast<const gdf_valid_type*>(column.valid)),
        identity(_identity) {}

  __host__ __device__ ResultType operator()(gdf_index_type i) const {
    //auto a = static_cast<double>(elements[i]);
    auto a = elements[i];
    return gdf_is_valid(bitmask, i)
               ? ((y >= a) ? Mypair(y-a,1) : Mypair(a-y,0))
               //? Mypair(abs(y - static_cast<double>(elements[i])),
               //         y >= static_cast<double>(elements[i]))
               : identity;
  }
};

template <typename T>
struct plusplus : public thrust::binary_function<T, T, T> {
  __host__ __device__ T operator()(const T& a, const T& b) {
    return T(a.first + b.first, a.second + b.second);
  }
};

template <typename T>
unsigned Objective(const gdf_column col, int n, double t, double* f,
                   double* df) {
  /* calculates the values of the objective and its subgradient */
  // abs_diff<T, Mypair> unary_op(t);
  plusplus<Mypair> binary_op;
  Mypair initpair(0.0, 0);
  auto transformer = abs_diff<T, Mypair>{col, Mypair(0.0, 0), t};
  // auto it = thrust::make_transform_iterator(
  //                               transformer);
  Mypair result = thrust::transform_reduce(
      thrust::counting_iterator<gdf_index_type>(0),
      thrust::counting_iterator<gdf_index_type>(col.size), transformer,
      initpair, binary_op);
  *df = 2.0 * result.second - n;
  *f = result.first;
  return result.second;
}

template <typename T>
struct inside_interval {
  const double L, R;
  T const* elements{};                    ///< pointer of cudf data array
  gdf_valid_type const* bitmask{};        ///< pointer of cudf bitmask (null) array
  inside_interval(const gdf_column column, const double& L, const double& R)
      : L(L),
        R(R),
        elements(static_cast<const T*>(column.data)),
        bitmask(reinterpret_cast<const gdf_valid_type*>(column.valid)) {}
  __host__ __device__ bool operator()(const gdf_index_type& i) {
    return gdf_is_valid(bitmask, i)
               ? (L < elements[i] && elements[i] < R)
               : false;
    // return (L < a && a < R);
  }
};

template <typename T>
double SortZ(const gdf_column col, double L, double R, int index,
             unsigned interval_size)
/* Copies the data satisfying L<data[i]<R into Z and returns the n/2-index
   order statistic of Z after sorting. */
{
  std::cout<<"L="<<L<<",R="<<R<<",index="<<index<<",interval_size="<<interval_size<<std::endl;
  inside_interval<T> pred{col, L, R};
  thrust::device_vector<T> Z(interval_size);

  auto endZ = thrust::copy_if(
      thrust::counting_iterator<gdf_index_type>(0),
      thrust::counting_iterator<gdf_index_type>(col.size), Z.begin(), pred);
  thrust::sort(Z.begin(), endZ);
  std::cout<<"zn="<<interval_size<<"ez="<<thrust::distance(Z.begin(), endZ)<<std::endl;
  auto n = col.size - col.null_count;
  //auto kth = llroundf(n / 2.0) - index;
  auto kth = n / 2 - index;
  std::cout<<"[n/2]-i="<<kth<<",index="<<index<<std::endl;
  if (n%2 == 0)
    return (Z[kth] + Z[kth-1]) / 2.0;
  else
    return Z[kth];
  // TODO return float/double/T ? gdf_scalar? union?
}

namespace detail {
struct median {
  template <typename T,
            typename std::enable_if_t<!std::is_arithmetic<T>::value>* = nullptr>
  gdf_scalar operator()(const gdf_column col) {
    CUDF_FAIL("input data type is not convertible to output data type");
  }
  template <typename T,
            typename std::enable_if_t<std::is_arithmetic<T>::value>* = nullptr>
  gdf_scalar operator()(const gdf_column col) {
    //TODO: fix this
    //T yL = reinterpret_cast<T>(cudf::reduction::min(col, col.dtype).data);
    //T yR = reinterpret_cast<T>(cudf::reduction::max(col, col.dtype).data);
    double yL = cudf::reduction::min(col, GDF_FLOAT64).data.fp64;
    double yR = cudf::reduction::max(col, GDF_FLOAT64).data.fp64;
    std::cout<<"min: "<<yL<<std::endl;
    std::cout<<"max: "<<yR<<std::endl;
    auto n = col.size - col.null_count;
    decltype(col.size) ltL = 1, ltR = n;
    std::cout<<"ltL="<<ltL
    <<"ltR="<<ltR
    <<std::endl;
    double fL, gL, fR, gR;
    gL = -n + 2;
    gR = n - 2; //wrong: as per paper
    ltL = Objective<T>(col, n, yL, &fL, &gL);
    ltR = Objective<T>(col, n, yR, &fR, &gR);
    std::cout<<"ltL="<<ltL<<",lTR="<<ltR<<std::endl;
    // TODO: need to make sure no overflow
    double sum = cudf::reduction::sum(col, GDF_FLOAT64).data.fp64;
    std::cout<<"sum: "<<sum<<std::endl;
    fL = sum - n * yL;
    fR = n * yR - sum;
    double t = NAN;
    // an approximate solution y, under 30 iterations with n up to 32 million
    // and tolerancef = 10-12.
    for (int i = 0; i < 7; i++) {
      t = (fR - fL + yL * gL - yR * gR) / (gL - gR);
    std::cout<<" fL="<<fL
             <<",fR="<<fR
             <<",gL="<<gL
             <<",gR="<<gR
             <<",yL="<<yL
             <<",yR="<<yR
             <<",t="<<t;
      double ft, gt;
      unsigned ltt = Objective<T>(col, n, t, &ft, &gt);
      if (gt < 0) {
        yL = t;
        fL = ft;
        gL = gt;
        ltL = ltt;
      } else {
        yR = t;
        fR = ft;
        gR = gt;
        ltR = ltt;
      }
    std::cout<<",ft="<<ft
             <<",gt="<<gt
             <<",lt="<<ltt
             <<std::endl;
      //std::cout<<(ltR-ltL)<<",";
      //// stop if interval has <1M elements
      if ((ltR - ltL) < (1LL<<20)) {
        std::cout<<"break-"<<i<<std::endl;
        break;
      }
    }
 
    gdf_scalar result{.data={.fp64=NAN}, .dtype=GDF_FLOAT64, .is_valid=true};
    result.data.fp64 = SortZ<T>(col, yL, yR, ltL, ltR - ltL);
    std::cout<<"\nres="<<result.data.fp64<<std::endl;
    return result;
  }
};
// template double median<double>(const gdf_column col);
}  // namespace detail

gdf_scalar median(const gdf_column col) {
  return cudf::type_dispatcher(col.dtype, detail::median(), col);
}

