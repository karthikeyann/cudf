#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "nested_json.hpp"

#include <hash/hash_allocator.cuh>
#include <hash/helper_functions.cuh>
#include <io/utilities/hostdevice_vector.hpp>

#include <cudf/detail/hashing.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/scatter.cuh>
#include <cudf/detail/utilities/hash_functions.cuh>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/utilities/span.hpp>

#include <cuco/static_map.cuh>

#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>
#include <rmm/mr/device/polymorphic_allocator.hpp>

#include <thrust/copy.h>
#include <thrust/gather.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_output_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/scan.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/tabulate.h>
#include <thrust/transform.h>
#include <thrust/uninitialized_fill.h>

#include <limits>

namespace cudf::io::json {
namespace detail {

// DEBUG print
template <typename T>
void print_vec(T const& cpu, std::string const name)
{
  for (auto const& v : cpu)
    printf("%3d,", int(v));
  std::cout << name << std::endl;
}

// The node that a token represents
struct token_to_node {
  __device__ auto operator()(PdaTokenT const token) -> NodeT
  {
    switch (token) {
      case token_t::StructBegin: return NC_STRUCT;
      case token_t::ListBegin: return NC_LIST;
      case token_t::StringBegin: return NC_STR;
      case token_t::ValueBegin: return NC_VAL;
      case token_t::FieldNameBegin: return NC_FN;
      default: return NC_ERR;
    };
  }
};

// Convert token indices to node range for each valid node.
template <typename T1, typename T2, typename T3>
struct node_ranges {
  T1 tokens;
  T2 token_indices;
  T3 num_tokens;
  bool include_quote_char;
  __device__ auto operator()(size_type i) -> thrust::tuple<SymbolOffsetT, SymbolOffsetT>
  {
    // Whether a token expects to be followed by its respective end-of-* token partner
    auto is_begin_of_section = [] __device__(PdaTokenT const token) {
      switch (token) {
        case token_t::StringBegin:
        case token_t::ValueBegin:
        case token_t::FieldNameBegin: return true;
        default: return false;
      };
    };
    // The end-of-* partner token for a given beginning-of-* token
    auto end_of_partner = [] __device__(PdaTokenT const token) {
      switch (token) {
        case token_t::StringBegin: return token_t::StringEnd;
        case token_t::ValueBegin: return token_t::ValueEnd;
        case token_t::FieldNameBegin: return token_t::FieldNameEnd;
        default: return token_t::ErrorBegin;
      };
    };
    // Includes quote char for end-of-string token or Skips the quote char for
    // beginning-of-field-name token
    auto get_token_index = [include_quote_char = include_quote_char] __device__(
                             PdaTokenT const token, SymbolOffsetT const token_index) {
      constexpr SymbolOffsetT quote_char_size = 1;
      switch (token) {
        // Strip off quote char included for StringBegin
        case token_t::StringBegin: return token_index + (include_quote_char ? 0 : quote_char_size);
        // Strip off or Include trailing quote char for string values for StringEnd
        case token_t::StringEnd: return token_index + (include_quote_char ? quote_char_size : 0);
        // Strip off quote char included for FieldNameBegin
        case token_t::FieldNameBegin: return token_index + quote_char_size;
        default: return token_index;
      };
    };
    PdaTokenT const token = tokens[i];
    // The section from the original JSON input that this token demarcates
    SymbolOffsetT range_begin = get_token_index(token, token_indices[i]);
    SymbolOffsetT range_end   = range_begin + 1;
    if (is_begin_of_section(token)) {
      if ((i + 1) < num_tokens && end_of_partner(token) == tokens[i + 1]) {
        // Update the range_end for this pair of tokens
        range_end = token_indices[i + 1];
      }
    }
    return thrust::make_tuple(range_begin, range_end);
  }
};

// Generates a tree representation of the given tokens, token_indices.
tree_meta_t get_tree_representation(device_span<PdaTokenT const> tokens,
                                    device_span<SymbolOffsetT const> token_indices,
                                    rmm::cuda_stream_view stream,
                                    rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  // Whether a token does represent a node in the tree representation
  auto is_node = [] __device__(PdaTokenT const token) -> size_type {
    switch (token) {
      case token_t::StructBegin:
      case token_t::ListBegin:
      case token_t::StringBegin:
      case token_t::ValueBegin:
      case token_t::FieldNameBegin:
      case token_t::ErrorBegin: return 1;
      default: return 0;
    };
  };

  // Whether the token pops from the parent node stack
  auto does_pop = [] __device__(PdaTokenT const token) {
    switch (token) {
      case token_t::StructMemberEnd:
      case token_t::StructEnd:
      case token_t::ListEnd: return true;
      default: return false;
    };
  };

  // Whether the token pushes onto the parent node stack
  auto does_push = [] __device__(PdaTokenT const token) {
    switch (token) {
      case token_t::FieldNameBegin:
      case token_t::StructBegin:
      case token_t::ListBegin: return true;
      default: return false;
    };
  };

  auto num_tokens = tokens.size();
  auto is_node_it = thrust::make_transform_iterator(tokens.begin(), is_node);
  auto num_nodes  = thrust::reduce(rmm::exec_policy(stream), is_node_it, is_node_it + num_tokens);

  // Node categories: copy_if with transform.
  rmm::device_uvector<NodeT> node_categories(num_nodes, stream, mr);
  auto node_categories_it =
    thrust::make_transform_output_iterator(node_categories.begin(), token_to_node{});
  auto node_categories_end = thrust::copy_if(rmm::exec_policy(stream),
                                             tokens.begin(),
                                             tokens.begin() + num_tokens,
                                             node_categories_it,
                                             is_node);
  CUDF_EXPECTS(node_categories_end - node_categories_it == num_nodes,
               "node category count mismatch");

  // Node levels: transform_exclusive_scan, copy_if.
  rmm::device_uvector<size_type> token_levels(num_tokens, stream);
  auto push_pop_it = thrust::make_transform_iterator(
    tokens.begin(), [does_push, does_pop] __device__(PdaTokenT const token) -> size_type {
      return does_push(token) ? 1 : (does_pop(token) ? -1 : 0);
    });
  thrust::exclusive_scan(
    rmm::exec_policy(stream), push_pop_it, push_pop_it + num_tokens, token_levels.begin());

  rmm::device_uvector<TreeDepthT> node_levels(num_nodes, stream, mr);
  auto node_levels_end = thrust::copy_if(rmm::exec_policy(stream),
                                         token_levels.begin(),
                                         token_levels.begin() + num_tokens,
                                         tokens.begin(),
                                         node_levels.begin(),
                                         is_node);
  CUDF_EXPECTS(node_levels_end - node_levels.begin() == num_nodes, "node level count mismatch");

  // Node ranges: copy_if with transform.
  rmm::device_uvector<SymbolOffsetT> node_range_begin(num_nodes, stream, mr);
  rmm::device_uvector<SymbolOffsetT> node_range_end(num_nodes, stream, mr);
  auto node_range_tuple_it =
    thrust::make_zip_iterator(node_range_begin.begin(), node_range_end.begin());
  // Whether the tokenizer stage should keep quote characters for string values
  // If the tokenizer keeps the quote characters, they may be stripped during type casting
  constexpr bool include_quote_char = true;
  using node_ranges_t =
    node_ranges<decltype(tokens.begin()), decltype(token_indices.begin()), decltype(num_tokens)>;
  auto node_range_out_it = thrust::make_transform_output_iterator(
    node_range_tuple_it,
    node_ranges_t{tokens.begin(), token_indices.begin(), num_tokens, include_quote_char});

  auto node_range_out_end =
    thrust::copy_if(rmm::exec_policy(stream),
                    thrust::make_counting_iterator<size_type>(0),
                    thrust::make_counting_iterator<size_type>(0) + num_tokens,
                    node_range_out_it,
                    [is_node, tokens_gpu = tokens.begin()] __device__(size_type i) -> bool {
                      PdaTokenT const token = tokens_gpu[i];
                      return is_node(token);
                    });
  CUDF_EXPECTS(node_range_out_end - node_range_out_it == num_nodes, "node range count mismatch");

  // Node parent ids: previous push token_id transform, stable sort, segmented scan with Max,
  // reorder, copy_if. This one is sort of logical stack. But more generalized.
  // TODO: make it own function.
  rmm::device_uvector<size_type> parent_token_ids(num_tokens, stream);
  rmm::device_uvector<size_type> initial_order(num_tokens, stream);
  thrust::sequence(rmm::exec_policy(stream), initial_order.begin(), initial_order.end());
  thrust::tabulate(rmm::exec_policy(stream),
                   parent_token_ids.begin(),
                   parent_token_ids.end(),
                   [does_push, tokens_gpu = tokens.begin()] __device__(auto i) -> size_type {
                     if (i == 0)
                       return -1;
                     else
                       return does_push(tokens_gpu[i - 1]) ? i - 1 : -1;
                   });
  auto out_pid = thrust::make_zip_iterator(parent_token_ids.data(), initial_order.data());
  // Uses radix sort for builtin types.
  thrust::stable_sort_by_key(rmm::exec_policy(stream),
                             token_levels.data(),
                             token_levels.data() + token_levels.size(),
                             out_pid);
  // SegmentedScan Max.
  thrust::inclusive_scan_by_key(rmm::exec_policy(stream),
                                token_levels.data(),
                                token_levels.data() + token_levels.size(),
                                parent_token_ids.data(),
                                parent_token_ids.data(),  // size_type{-1},
                                thrust::equal_to<size_type>{},
                                thrust::maximum<size_type>{});
  // FIXME: Avoid sorting again by scatter + extra memory, or permutation iterator for
  // parent_token_ids. Tradeoff?
  thrust::sort_by_key(rmm::exec_policy(stream),
                      initial_order.data(),
                      initial_order.data() + initial_order.size(),
                      parent_token_ids.data());
  // thrust::scatter(rmm::exec_policy(stream),
  //                parent_token_ids.begin(),
  //                parent_token_ids.end(),
  //                initial_order.data(),
  //                parent_token_ids.begin()); //same location not allowed in scatter
  rmm::device_uvector<size_type> node_ids_gpu(num_tokens, stream);
  thrust::exclusive_scan(
    rmm::exec_policy(stream), is_node_it, is_node_it + num_tokens, node_ids_gpu.begin());
  rmm::device_uvector<NodeIndexT> parent_node_ids(num_nodes, stream, mr);
  auto parent_node_ids_it = thrust::make_transform_iterator(
    parent_token_ids.begin(),
    [node_ids_gpu = node_ids_gpu.begin()] __device__(size_type const pid) -> NodeIndexT {
      return pid < 0 ? pid : node_ids_gpu[pid];
    });
  auto parent_node_ids_end = thrust::copy_if(rmm::exec_policy(stream),
                                             parent_node_ids_it,
                                             parent_node_ids_it + parent_token_ids.size(),
                                             tokens.begin(),
                                             parent_node_ids.begin(),
                                             is_node);
  CUDF_EXPECTS(parent_node_ids_end - parent_node_ids.begin() == num_nodes,
               "parent node id gather mismatch");
  return {std::move(node_categories),
          std::move(parent_node_ids),
          std::move(node_levels),
          std::move(node_range_begin),
          std::move(node_range_end)};
}

// JSON tree traversal for record orient. (list of structs)
// returns col_id of each node, and row_offset(TODO)
void records_orient_tree_traversal(device_span<SymbolT const> d_input,
                                   tree_meta_t& d_tree,
                                   rmm::cuda_stream_view stream,
                                   rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  // GPU version
  // 3. convert node_category+fieldname to node_type!
  using hash_table_allocator_type = rmm::mr::stream_allocator_adaptor<default_allocator<char>>;
  using hash_map_type =
    cuco::static_map<size_type, size_type, cuda::thread_scope_device, hash_table_allocator_type>;

  constexpr size_type empty_node_index_sentinel = std::numeric_limits<size_type>::max();
  auto num_nodes                                = d_tree.node_categories.size();
  hash_map_type key_map{compute_hash_table_size(num_nodes),
                        cuco::sentinel::empty_key{empty_node_index_sentinel},
                        cuco::sentinel::empty_value{empty_node_index_sentinel},
                        hash_table_allocator_type{default_allocator<char>{}, stream},
                        stream.value()};
  auto d_hasher = [d_input          = d_input.data(),
                   node_range_begin = d_tree.node_range_begin.data(),
                   node_range_end   = d_tree.node_range_end.data()] __device__(auto node_id) {
    // TODO if node_category is hashed, then no need for transform later. for field only, use string
    // hash too.
    auto field_name = cudf::string_view(d_input + node_range_begin[node_id],
                                        node_range_end[node_id] - node_range_begin[node_id]);
    return cudf::detail::default_hash<cudf::string_view>{}(field_name);
  };
  auto d_equal = [d_input          = d_input.data(),
                  node_range_begin = d_tree.node_range_begin.data(),
                  node_range_end   = d_tree.node_range_end.data()] __device__(auto node_id1,
                                                                            auto node_id2) {
    // TODO if node_category is used, then no need for transform later.
    auto field_name1 = cudf::string_view(d_input + node_range_begin[node_id1],
                                         node_range_end[node_id1] - node_range_begin[node_id1]);
    auto field_name2 = cudf::string_view(d_input + node_range_begin[node_id2],
                                         node_range_end[node_id2] - node_range_begin[node_id2]);
    return field_name1 == field_name2;
  };
  auto is_field_node = [node_categories = d_tree.node_categories.data()] __device__(auto node_id) {
    return node_categories[node_id] == node_t::NC_FN;
  };
  // key-value pairs: uses node_id itself as node_type. (unique node_id for a field name due to
  // hashing)
  auto iter = cudf::detail::make_counting_transform_iterator(
    0, [] __device__(size_type i) { return cuco::make_pair(i, i); });

  key_map.insert_if(iter,
                    iter + num_nodes,
                    thrust::counting_iterator<size_type>(0),  // stencil
                    is_field_node,
                    d_hasher,
                    d_equal,
                    stream.value());
  auto get_hash_value =
    [key_map = key_map.get_device_view(), d_hasher, d_equal] __device__(auto node_id) -> size_type {
    auto it = key_map.find(node_id, d_hasher, d_equal);
    return (it == key_map.end()) ? size_type{0} : it->second.load();
  };
  // convert field nodes to node indices, and other nodes to enum value.
  rmm::device_uvector<size_type> node_type(num_nodes, stream);
  thrust::tabulate(rmm::exec_policy(stream),
                   node_type.begin(),
                   node_type.end(),
                   [node_categories = d_tree.node_categories.data(),
                    is_field_node,
                    get_hash_value] __device__(auto node_id) -> size_type {
                     if (is_field_node(node_id))
                       return static_cast<size_type>(NUM_NODE_CLASSES) + get_hash_value(node_id);
                     else
                       return static_cast<size_type>(node_categories[node_id]);
                   });
  print_vec(cudf::detail::make_std_vector_async(node_type, stream), "node_type");
  // 1. Preprocessing: Translate parent node ids after sorting by level.
  //   a. sort by level
  //   b. get gather map of sorted indices
  //   c. translate parent_node_ids to sorted indices
  rmm::device_uvector<size_type> scatter_indices(num_nodes, stream);
  thrust::sequence(rmm::exec_policy(stream), scatter_indices.begin(), scatter_indices.end());
  printf("\n");
  print_vec(cudf::detail::make_std_vector_async(scatter_indices, stream), "gpu.node_id");
  print_vec(cudf::detail::make_std_vector_async(d_tree.parent_node_ids, stream),
            "gpu.parent_node_ids");
  print_vec(cudf::detail::make_std_vector_async(node_type, stream), "gpu.node_type");
  print_vec(cudf::detail::make_std_vector_async(d_tree.node_levels, stream), "gpu.node_levels");
  auto out_pid = thrust::make_zip_iterator(scatter_indices.data(),
                                           //  d_tree.node_levels.data(),
                                           d_tree.parent_node_ids.data(),
                                           node_type.data());
  //  d_tree.node_categories.data());
  // TODO: use cub radix sort.
  thrust::stable_sort_by_key(rmm::exec_policy(stream),
                             d_tree.node_levels.data(),
                             d_tree.node_levels.data() + num_nodes,
                             out_pid);
  auto gather_indices = cudf::detail::scatter_to_gather(
    scatter_indices.begin(), scatter_indices.end(), num_nodes, stream);

  rmm::device_uvector<NodeIndexT> parent_indices(num_nodes, stream);
  *thrust::device_pointer_cast(parent_indices.data()) = -1;
  thrust::gather(rmm::exec_policy(stream),
                 d_tree.parent_node_ids.begin() + 1,  // first node's parent is -1
                 d_tree.parent_node_ids.end(),
                 gather_indices.begin(),
                 parent_indices.begin() + 1);
  printf("\n");
  print_vec(cudf::detail::make_std_vector_async(scatter_indices, stream), "gpu.node_id");
  print_vec(cudf::detail::make_std_vector_async(d_tree.parent_node_ids, stream),
            "gpu.parent_node_ids");
  print_vec(cudf::detail::make_std_vector_async(node_type, stream), "gpu.node_type");
  print_vec(cudf::detail::make_std_vector_async(d_tree.node_levels, stream), "gpu.node_levels");
  print_vec(cudf::detail::make_std_vector_async(gather_indices, stream), "new_home");
  print_vec(cudf::detail::make_std_vector_async(parent_indices, stream), "parent_indices");
  // XXX: restore parent_node_ids order using scatter. (check if this order is right?)
  rmm::device_uvector<NodeIndexT> parent_node_ids(num_nodes, stream);  // Used later for row_offsets
  thrust::scatter(rmm::exec_policy(stream),
                  d_tree.parent_node_ids.begin(),
                  d_tree.parent_node_ids.end(),
                  scatter_indices.begin(),
                  parent_node_ids.begin());
  print_vec(cudf::detail::make_std_vector_async(parent_node_ids, stream),
            "parent_node_ids (restored)");
  // 2. Find level boundaries.
  hostdevice_vector<size_type> level_boundaries(num_nodes + 1, stream);
  auto level_end = thrust::copy_if(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator<size_type>(1),
    thrust::make_counting_iterator<size_type>(num_nodes + 1),
    level_boundaries.d_begin(),
    [num_nodes, node_levels = d_tree.node_levels.begin()] __device__(auto index) {
      return index == 0 || index == num_nodes || node_levels[index] != node_levels[index - 1];
    });
  level_boundaries.device_to_host(stream, true);
  print_vec(level_boundaries, "level_boundaries");
  auto num_levels = level_end - level_boundaries.d_begin();
  std::cout << "num_levels: " << num_levels << std::endl;
  // level_boundaries[num_levels] = num_nodes;

  auto print_level_data = [stream](auto level,
                                   auto start,
                                   auto end,
                                   auto const& nodeII,
                                   auto const& parent_node_idx,
                                   auto const& parent_col_id,
                                   auto const& node_type,
                                   auto const& levels,
                                   auto const& col_id) {
    auto cls = std::array<const char*, 6>{"S", "L", "F", "R", "V", "E"};
    for (auto n = start; n < end; n++)
      printf("%3d ", nodeII.element(n, stream));
    printf(" nodeII-%ld\n", level);
    for (auto n = start; n < end; n++)
      printf("%3d ", parent_node_idx.element(n, stream));
    printf(" parent_node_idx-%ld\n", level);
    for (auto n = start; n < end; n++)
      printf("%3d ", parent_col_id.element(n, stream));
    printf(" parent_col_id-%ld\n", level);
    for (auto n = start; n < end; n++) {
      auto nt = node_type.element(n, stream);
      printf("%3s ",
             nt >= NUM_NODE_CLASSES ? std::to_string(nt - NUM_NODE_CLASSES).c_str() : cls[nt]);
    }
    printf(" node_type-%ld\n", level);
    for (auto n = start; n < end; n++)
      printf("%3d ", levels.element(n, stream));
    printf(" levels-%ld\n", level);
    for (auto n = start; n < end; n++)
      printf("%3d ", col_id.element(n, stream));
    printf(" col_id-%ld\n", level);
  };

  // 4. Propagate parent node ids for each level.
  // For each level,
  //     a. gather col_id from previous level results. input=col_id, gather_map is parent_indices.
  //     b. sort by {col_id, type}
  //     c. scan sum of unique {parent_col_id, type}
  // cross check outputs.
  // Calculate row offsets too.
  rmm::device_uvector<size_type> col_id(num_nodes, stream);
  rmm::device_uvector<size_type> parent_col_id(num_nodes, stream);
  thrust::uninitialized_fill(rmm::exec_policy(stream),
                             parent_col_id.begin(),
                             parent_col_id.end(),
                             0);  // XXX: is this needed?
  thrust::uninitialized_fill(rmm::exec_policy(stream), col_id.begin(), col_id.end(), 0);  ///
  thrust::device_pointer_cast(parent_col_id.data())[0] = -1;
  for (decltype(num_levels) level = 1; level < num_levels; level++) {
    // std::cout << level << ".before gather\n";
    thrust::gather(rmm::exec_policy(stream),
                   parent_indices.data() +
                     level_boundaries[level - 1],  // FIXME: might be wrong. might be a bug here.
                   parent_indices.data() + level_boundaries[level],
                   col_id.data(),  // + level_boundaries[level - 1],
                   parent_col_id.data() + level_boundaries[level - 1]);
    // std::cout << level << ".after gather\n";
    // print_level_data(level,
    //                  level_boundaries[level - 1],
    //                  level_boundaries[level],
    //                  scatter_indices,
    //                  parent_indices,
    //                  parent_col_id,
    //                  node_type,
    //                  d_tree.node_levels,
    //                  col_id);
    // std::cout << level << ".before sort\n";
    // TODO probably sort_by_key value should be a gather/scatter index to restore original order.
    thrust::stable_sort_by_key(
      rmm::exec_policy(stream),
      thrust::make_zip_iterator(parent_col_id.begin() + level_boundaries[level - 1],
                                node_type.data() + level_boundaries[level - 1]),
      thrust::make_zip_iterator(parent_col_id.begin() + level_boundaries[level],
                                node_type.data() + level_boundaries[level]),
      thrust::make_zip_iterator(
        scatter_indices.begin() +
        level_boundaries[level - 1]  //, // is this required?
                                     //  gather_indices.begin() + level_boundaries[level - 1],
                                     //  parent_indices.begin() + level_boundaries[level - 1]
        ));
    // std::cout << level << ".after sort\n";
    // print_level_data(level,
    //                  level_boundaries[level - 1],
    //                  level_boundaries[level],
    //                  scatter_indices,
    //                  parent_indices,
    //                  parent_col_id,
    //                  node_type,
    //                  d_tree.node_levels,
    //                  col_id);
    auto start_it = thrust::make_zip_iterator(parent_col_id.begin() + level_boundaries[level - 1],
                                              node_type.data() + level_boundaries[level - 1]);
    auto adjacent_pair_it = thrust::make_zip_iterator(start_it - 1, start_it);
    // std::cout << level << ".before transform\n";
    thrust::transform(rmm::exec_policy(stream),
                      adjacent_pair_it,
                      adjacent_pair_it + level_boundaries[level] - level_boundaries[level - 1],
                      col_id.data() + level_boundaries[level - 1],
                      [] __device__(auto adjacent_pair) -> size_type {
                        auto lhs = thrust::get<0>(adjacent_pair),
                             rhs = thrust::get<1>(adjacent_pair);
                        return lhs != rhs ? 1 : 0;
                      });
    // std::cout << level << ".before scan\n";
    // // includes previous level last col_id to continue the index.
    thrust::inclusive_scan(rmm::exec_policy(stream),
                           col_id.data() + level_boundaries[level - 1] - 1,
                           col_id.data() + level_boundaries[level],
                           col_id.data() + level_boundaries[level - 1] - 1);
    // // print node_id, parent_node_idx, parent_col_id, node_type, level.
    // std::cout << level << ".after scan\n";
    // print_level_data(level,
    //                  level_boundaries[level - 1],
    //                  level_boundaries[level],
    //                  scatter_indices,
    //                  parent_indices,
    //                  parent_col_id,
    //                  node_type,
    //                  d_tree.node_levels,
    //                  col_id);
    // TODO scatter/gather to restore original order. (scatter will be faster.)
    thrust::sort_by_key(
      rmm::exec_policy(stream),
      scatter_indices.begin() + level_boundaries[level - 1],
      scatter_indices.begin() + level_boundaries[level],
      thrust::make_zip_iterator(col_id.begin() + level_boundaries[level - 1],
                                parent_col_id.data() + level_boundaries[level - 1]));
    // print_level_data(level,
    //                  level_boundaries[level - 1],
    //                  level_boundaries[level],
    //                  scatter_indices,
    //                  parent_indices,
    //                  parent_col_id,
    //                  node_type,
    //                  d_tree.node_levels,
    //                  col_id);
  }
  // FIXME: to make parent_col_id of last level correct, do we need a gather here?
  thrust::gather(rmm::exec_policy(stream),
                 parent_indices.begin() +
                   level_boundaries[num_levels - 1],  // FIXME: might be wrong. might be a bug here.
                 parent_indices.end(),
                 col_id.data(),  // + level_boundaries[level - 1],
                 parent_col_id.data() + level_boundaries[num_levels - 1]);
  auto translate_col_id = [](auto col_id) {
    std::unordered_map<int, int> col_id_map;
    std::vector<int> new_col_ids(col_id.size());
    int unique_id = 0;
    for (auto id : col_id) {
      if (col_id_map.count(id) == 0) { col_id_map[id] = unique_id++; }
    }
    for (size_t i = 0; i < col_id.size(); i++) {
      new_col_ids[i] = col_id_map[col_id[i]];
    }
    return new_col_ids;
  };
  // restore original order of col_id.
  // TODO can we do this with scatter instead of sort?
  thrust::sort_by_key(rmm::exec_policy(stream),
                      scatter_indices.begin(),
                      scatter_indices.end(),
                      thrust::make_zip_iterator(parent_indices.begin(),
                                                node_type.begin(),
                                                parent_col_id.begin(),
                                                col_id.begin(),
                                                d_tree.node_levels.begin()));
  print_vec(cudf::detail::make_std_vector_async(scatter_indices, stream), "gpu.node_id");
  print_vec(cudf::detail::make_std_vector_async(parent_indices, stream),
            "gpu.parent_indices");  // once original order is restored, is this required?
  print_vec(cudf::detail::make_std_vector_async(node_type, stream),
            "gpu.node_type");  // is this needed?
  print_vec(cudf::detail::make_std_vector_async(parent_col_id, stream),
            "parent_col_id");                                                // is this needed?
  print_vec(cudf::detail::make_std_vector_async(col_id, stream), "col_id");  // required.
  print_vec(translate_col_id(cudf::detail::make_std_vector_async(col_id, stream)),
            "col_id (translated)");  // is this required? required to be ordered for the next step?
  print_vec(cudf::detail::make_std_vector_async(d_tree.node_levels, stream), "gpu.node_levels");
  // auto sorted_cpu_col_id = [&]() {
  //     auto sc = cudf::detail::make_std_vector_async(scatter_indices, stream);
  //     std::vector<size_type> sorted_cpu_col_id(sc.size());
  //     for(decltype(sc.size()) i=0; i<sc.size(); i++) {
  //       sorted_cpu_col_id[sc[i]] = node_ids[i];
  //     }
  //     return sorted_cpu_col_id;
  // }();
  // print_vec(sorted_cpu_col_id, "cpu.node_id (sorted)");

  // auto sc = cudf::detail::make_std_vector_async(scatter_indices, stream);
  // for(int i=0; i< int(cpu_tree.node_range_begin.size()); i++) {
  //   printf("%3s ", std::string(input.data() + cpu_tree.node_range_begin[sc[i]],
  //   cpu_tree.node_range_end[sc[i]] - cpu_tree.node_range_begin[sc[i]]).c_str());
  // }
  // printf(" (JSON)\n");

  // 5. Generate row_offset.
  // stable_sort by parent_col_id.
  // scan_by_key on nodes who's parent is list on col_id.
  // propagate to leaves! how?
  thrust::stable_sort_by_key(
    rmm::exec_policy(stream), parent_col_id.begin(), parent_col_id.end(), scatter_indices.begin());
  rmm::device_uvector<size_type> row_offsets(num_nodes, stream);
  // TODO is it possible to generate list child_offsets too here?
  thrust::exclusive_scan_by_key(
    rmm::exec_policy(stream),
    parent_col_id.begin(),  // TODO: is there any way to limit this to list parents alone?
    parent_col_id.end(),
    thrust::make_constant_iterator<size_type>(1),
    row_offsets.begin());
  print_vec(cudf::detail::make_std_vector_async(parent_col_id, stream), "parent_col_id");
  print_vec(cudf::detail::make_std_vector_async(row_offsets, stream), "row_offsets (generated)");
  thrust::sort_by_key(rmm::exec_policy(stream),
                      scatter_indices.begin(),
                      scatter_indices.end(),
                      thrust::make_zip_iterator(parent_col_id.begin(), row_offsets.begin()));
  thrust::transform_if(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator<size_type>(0),
    thrust::make_counting_iterator<size_type>(num_nodes),
    row_offsets.begin(),
    [node_categories = d_tree.node_categories.data(),
     parent_node_ids = parent_node_ids.begin(),
     row_offsets     = row_offsets.begin()] __device__(size_type node_id) {
      auto parent_node_id = parent_node_ids[node_id];
      while (node_categories[parent_node_id] != node_t::NC_LIST &&
             parent_node_id != -1) {  // TODO replace -1 with sentinel
        node_id        = parent_node_id;
        parent_node_id = parent_node_ids[parent_node_id];
      }
      return row_offsets[node_id];
    },
    [node_categories = d_tree.node_categories.data(),
     parent_node_ids = parent_node_ids.begin()] __device__(size_type node_id) {
      auto parent_node_id = parent_node_ids[node_id];
      return parent_node_id != -1 and
             !(node_categories[parent_node_id] ==
               node_t::NC_LIST);  // Parent is not a list, or sentinel/root (might be different
                                  // condition for JSON_lines)
    });
  print_vec(cudf::detail::make_std_vector_async(row_offsets, stream), "row_offsets (generated)");
  // For now: simple while loop for each thread to retrieve parents row_offset until a node's parent
  // is list node. thrust::transform(rmm::exec_policy(stream), //parent node_id, node_category.
  // problem with using parent_col_id is that it may not be null literal. scan operation is fine but
  // how? propagate to leaves in parallel? does it have to be done level by level? need not be
  // because there may be lists in between. revert back the order and simple scan_max is enough?
  // won't work. regardless of order, a simple scan of op(a,b): return if b==0? a: b; will work.
  // (need to be associative.)

  // TODO return col_id, row_offset of each node.
}

}  // namespace detail
}  // namespace cudf::io::json
